/**
* CUDA RENDER
*
* @author Sam Collier
*/

#include <hip/hip_runtime.h>
#include "math.h"

__device__
double dot3d(double x0,double y0,double z0,double x1,double y1,double z1)
{
	return x0*x1+y0*y1+z0*z1;
}
__device__
double cross3dX(double y0,double z0,double y1,double z1)
{
	return y0*z1-z0*y1;
}
__device__
double cross3dY(double x0,double z0,double x1,double z1)
{
	return z0*x1-x0*z1;
}
__device__
double cross3dZ(double x0,double y0,double x1,double y1)
{
	return x0*y1-y0*x1;
}
__device__
double imagePlanPoint(double camPos,double camForward,double camUp,
	double camRight,double u,double v,double xFoV,double yFoV)
{
	return camPos+camForward+camRight*2*(u-.5)*tan(xFoV)+
					camUp*2*(v-.5)*tan(yFoV);
}
extern "C"
__global__
void render(int n,int xRes,int yRes,
	double camX,double camY,double camZ,
	double camforX,double camforY,double camforZ,
	double camupX,double camupY,double camupZ,
	double fov,double aspectRatio,
	double* v0X,double* v0Y,double* v0Z,
	double* v1X,double* v1Y,double* v1Z,
	double* v2X,double* v2Y,double* v2Z,
	double meshposX,double meshposY,double meshposZ,
	double* outR,double* outG,double* outB)
{
	const int EPSILON=1e-6;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	int colStride = blockDim.x*gridDim.x;
	int row=blockIdx.y*blockDim.y+threadIdx.y;
	int rowStride=blockDim.y*gridDim.y;
	if(col<xRes&&row<yRes)
	{
		bool bIntersected=false;

		// Calculating right vector using cross product
		double rightX=cross3dX(camforY,camforZ,camupY,camupZ);
		double rightY=cross3dY(camforX,camforZ,camupX,camupZ);
		double rightZ=cross3dZ(camforX,camforY,camupX,camupY);
			
		double xFoV=fov*(acos(-1.0)/180); //convert from degree to radian
		double yFoV=atan(tan(xFoV)/aspectRatio); //getting yFoV from xFoV
		
		for(int i=col;i<xRes;i+=colStride)
		{
			for(int j=row;j<yRes;j+=rowStride)
			{
				double u=(i+.5)/xRes;
				double v=(j+.5)/yRes;
				

				double imagePlainPointX=imagePlanPoint(camX,camforX,
					camupX,rightX,u,v,xFoV,yFoV);
				double imagePlainPointY=imagePlanPoint(camY,camforY,
					camupY,rightY,u,v,xFoV,yFoV);
				double imagePlainPointZ=imagePlanPoint(camZ,camforZ,
					camupZ,rightZ,u,v,xFoV,yFoV);

				double reciprocalLength=rnorm3d(imagePlainPointX-camX,imagePlainPointY-camY,
					imagePlainPointZ-camZ);

				// Normalize each component
				double rayDirX=(imagePlainPointX-camX)*reciprocalLength;
				double rayDirY=(imagePlainPointY-camY)*reciprocalLength;
				double rayDirZ=(imagePlainPointZ-camZ)*reciprocalLength;

				// Check for intersection with every triangle in mesh
				for(int t=0;t<n;t++)
				{
					// v0, v1, v2 represent vertices
					// Adding the mesh position to all vertices
					// because the vertices' coords are relative
					// to the mesh, not the scene/world
					double meshV0X=v0X[t]+meshposX;
					double meshV0Y=v0Y[t]+meshposY;
					double meshV0Z=v0Z[t]+meshposZ;
					double meshV1X=v1X[t]+meshposX;
					double meshV1Y=v1Y[t]+meshposY;
					double meshV1Z=v1Z[t]+meshposZ;
					double meshV2X=v2X[t]+meshposX;
					double meshV2Y=v2Y[t]+meshposY;
					double meshV2Z=v2Z[t]+meshposZ;

					double aX=meshV1X-meshV0X;
					double aY=meshV1Y-meshV0Y;
					double aZ=meshV1Z-meshV0Z;
					double bX=meshV2X-meshV0X;
					double bY=meshV2Y-meshV0Y;
					double bZ=meshV2Z-meshV0Z;

					double cX=cross3dX(aY,aZ,bY,bZ);
					double cY=cross3dY(aX,aZ,bX,bZ);
					double cZ=cross3dZ(aX,aY,bX,bY);

					double normX=cX*rnorm3d(cX,cY,cZ);
					double normY=cY*rnorm3d(cX,cY,cZ);
					double normZ=cZ*rnorm3d(cX,cY,cZ);

					double d=dot3d(-meshV0X,-meshV0Y,-meshV0Z,
						normX,normY,normZ);
					
					double rayX=camX;
					double rayY=camY;
					double rayZ=camZ;
					double rayDotNorm=dot3d(rayX,rayY,rayZ,
						normX,normY,normZ);
					double rayDirDotNorm=dot3d(rayDirX,rayDirY,rayDirZ,
						normX,normY,normZ);
					
					double distance=-(rayDotNorm+d)/rayDirDotNorm;
					if(distance>EPSILON)
					{
						double interX = rayX+rayDirX*distance;
						double interY = rayY+rayDirY*distance;
						double interZ = rayZ+rayDirZ*distance;

						// calculations for a
						double eX=meshV1X-meshV0X;
						double eY=meshV1Y-meshV0Y;
						double eZ=meshV1Z-meshV0Z;
						double fX=interX-meshV0X;
						double fY=interY-meshV0Y;
						double fZ=interZ-meshV0Z;
						double gX=cross3dX(eY,eZ,fY,fZ);
						double gY=cross3dY(eX,eZ,fX,fZ);
						double gZ=cross3dZ(eX,eY,fX,fY);
						double a=dot3d(gX,gY,gZ,
							normX,normY,normZ);

						// calculations for b
						double hX=meshV2X-meshV1X;
						double hY=meshV2Y-meshV1Y;
						double hZ=meshV2Z-meshV1Z;
						double iX=interX-meshV1X;
						double iY=interY-meshV1Y;
						double iZ=interZ-meshV1Z;
						double jX=cross3dX(hY,hZ,iY,iZ);
						double jY=cross3dY(hX,hZ,iX,iZ);
						double jZ=cross3dZ(hX,hY,iX,iY);
						double b=dot3d(jX,jY,jZ,
							normX,normY,normZ);

						// calculations for c
						double kX=meshV0X-meshV2X;
						double kY=meshV0Y-meshV2Y;
						double kZ=meshV0Z-meshV2Z;
						double lX=interX-meshV2X;
						double lY=interY-meshV2Y;
						double lZ=interZ-meshV2Z;
						double mX=cross3dX(kY,kZ,lY,lZ);
						double mY=cross3dY(kX,kZ,lX,lZ);
						double mZ=cross3dZ(kX,kY,lX,lY);
						double c=dot3d(mX,mY,mZ,
							normX,normY,normZ);

						if(a>0&&b>0&&c>0)
						{
							bIntersected=true;
							break;
						}
					}
					if(bIntersected)break;
				}
				if(bIntersected)
				{
					outR[i+j*xRes]=237/255.0;
					outG[i+j*xRes]=142/255.0;
					outB[i+j*xRes]=0/255.0;
				}
			}
		}
	}
}